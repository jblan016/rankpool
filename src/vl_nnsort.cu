#include "hip/hip_runtime.h"
// @file vl_nnsort.cu
// @brief Sorting block MEX wrapper
// @author Andrea Vedaldi
// @author Karel Lenc
//modded by j.b.<2017>

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnsorting.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <assert.h>

/* option codes */
enum {
  opt_stride = 0,
  opt_pad,
  opt_method,
  opt_verbose
} ;

/* options */
VLMXOption  options [] = {
  {"Stride",           1,   opt_stride            },
  {"Pad",              1,   opt_pad               },
  {"Method",           1,   opt_method            },
  {"Verbose",          0,   opt_verbose           },
  {0,                  0,   0                     }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_SIZE, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  int sortWidth ;
  int sortHeight ;
  int strideX = 1 ;
  int strideY = 1 ;
  int padLeft = 0 ;
  int padRight = 0 ;
  int padTop = 0 ;
  int padBottom = 0 ;
  vl::SortingMethod method = vl::vlSortingMax ;
  bool backMode = false ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 2) {
    mexErrMsgTxt("The arguments are less than two.") ;
  }

  if (nin > 2 && vlmxIsString(in[2],-1)) {
    next = 2 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 3) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_stride :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("STRIDE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = strideY ;
            break ;
          case 2:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            mexErrMsgTxt("STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("PAD is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padLeft = (int)mxGetPr(optarg)[0] ;
            padRight = padLeft ;
            padTop = padLeft ;
            padBottom = padLeft ;
            break ;
          case 4:
            padTop = (int)mxGetPr(optarg)[0] ;
            padBottom = (int)mxGetPr(optarg)[1] ;
            padLeft = (int)mxGetPr(optarg)[2] ;
            padRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            mexErrMsgTxt("PAD has neither one nor four elements.") ;
        }
        break;

      case opt_method :
        if (!vlmxIsString(optarg,-1)) {
           vlmxError(VLMXE_IllegalArgument, "METHOD is not a string.") ;
        }
        if (vlmxIsEqualToStringI(optarg, "max")) {
          method = vl::vlSortingMax ;
        } else if (vlmxIsEqualToStringI(optarg, "avg")) {
          method = vl::vlSortingAverage ;
        } else {
          vlmxError(VLMXE_IllegalArgument, "METHOD is not a supported method.") ;
        }
        break;

      default:
        break ;
    }
  }
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(false) ;
#endif

  vl::MexTensor data(context) ;
  vl::MexTensor derOutput(context) ;

  data.init(in[IN_DATA]) ;
  data.reshape(4) ; // -> 4 dimensions

  if (backMode) {
    derOutput.init(in[IN_DEROUTPUT]) ;
    derOutput.reshape(4) ; // -> 4 dimensions
  }

  if (backMode && ! vl::areCompatible(data, derOutput)) {
    mexErrMsgTxt("DATA and DEROUTPUT do not have compatible formats.") ;
  }

  if (!vlmxIsPlainMatrix(in[IN_SIZE],-1,-1)) {
    mexErrMsgTxt("SIZE is not a plain matrix.") ;
  }
  switch (mxGetNumberOfElements(in[IN_SIZE])) {
    case 1:
      sortHeight = mxGetPr(in[IN_SIZE])[0] ;
      sortWidth = sortHeight ;
      break ;
    case 2:
      sortHeight = mxGetPr(in[IN_SIZE])[0] ;
      sortWidth = mxGetPr(in[IN_SIZE])[1] ;
      break ;
    default:
      mexErrMsgTxt("SIZE has neither one nor two elements.") ;
  }

  /* Basic compatibility of Shape */
  if (strideX < 1 || strideY < 1) {
    mexErrMsgTxt("At least one element of STRIDE is smaller than one.") ;
  }
  if (sortHeight == 0 || sortWidth == 0) {
    mexErrMsgTxt("A dimension of the sorting SIZE is void.") ;
  }
  if (data.getHeight() + (padTop+padBottom) < sortHeight ||
      data.getWidth() + (padLeft+padRight) < sortWidth) {
    mexErrMsgTxt("The sorting window is larger than the DATA (including padding).") ;
  }
  if (padLeft < 0 ||
      padRight < 0 ||
      padTop < 0 ||
      padBottom < 0) {
    mexErrMsgTxt("An element of PAD is negative.") ;
  }
  if (padLeft >= sortWidth ||
      padRight >= sortWidth ||
      padTop >= sortHeight  ||
      padBottom >= sortHeight) {
    mexErrMsgTxt("A padding value is larger or equal to the size of the sorting window.") ;
  }

  /* Get the output Shape */
  vl::TensorShape outputShape((data.getHeight() + (padTop+padBottom) - sortHeight)/strideY + 1,
                              (data.getWidth()  + (padLeft+padRight) - sortWidth)/strideX + 1,
                              data.getDepth(),
                              data.getSize()) ;

  if (backMode && (derOutput != outputShape)) {
    mexErrMsgTxt("DEROUTPUT dimensions are incompatible with X and SORT.") ;
  }

  /* Create output buffers */
  vl::DeviceType deviceType = data.getDeviceType() ;
  vl::DataType dataType = data.getDataType() ;
  vl::MexTensor output(context) ;
  vl::MexTensor derData(context) ;

  if (!backMode) {
    output.initWithZeros(deviceType, dataType, outputShape) ;
  } else {
    derData.initWithZeros(deviceType, dataType, data.getShape()) ;
  }

  if (verbosity > 0) {
    mexPrintf("vl_nnsort: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::VLDT_GPU) ? "GPU" : "CPU") ;
    if (data.getDeviceType() == vl::VLDT_GPU) {
#if ENABLE_CUDNN
      mexPrintf("; %s\n", context.getCudaHelper().getCudnnEnabled() ? "cuDNN" : "MatConvNet") ;
#else
      mexPrintf("; MatConvNet\n") ;
#endif
    } else {
      mexPrintf("; MatConvNet\n") ;
    }
    mexPrintf("vl_nnsort: stride: [%d %d], pad: [%d %d %d %d]\n",
              strideY, strideX,
              padTop, padBottom, padLeft, padRight) ;
    vl::print("vl_nnsort: data: ", data) ;
    mexPrintf("vl_nnsort: sorting: %d x %d\n", sortHeight, sortWidth);
    mexPrintf("vl_nnsort: method: %s\n", (method == vl::vlSortingMax) ? "max" : "avg") ;
    if (backMode) {
      vl::print("vl_nnsort: derOutput: ", derOutput) ;
      vl::print("vl_nnsort: derData: ", derData) ;
    } else {
      vl::print("vl_nnsort: output: ", output) ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  vl::ErrorCode error ;
  if (!backMode) {
    error = vl::nnsorting_forward(context,
                                  output, data,
                                  method,
                                  sortHeight, sortWidth,
                                  strideY, strideX,
                                  padTop, padBottom, padLeft, padRight) ;
  } else {
    error = vl::nnsorting_backward(context,
                                   derData, data, derOutput,
                                   method,
                                   sortHeight, sortWidth,
                                   strideY, strideX,
                                   padTop, padBottom, padLeft, padRight) ;
  }

  /* -------------------------------------------------------------- */
  /*                                                         Finish */
  /* -------------------------------------------------------------- */

  if (error != vl::VLE_Success) {
    mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
  }
  if (backMode) {
    out[OUT_RESULT] = derData.relinquish() ;
  } else {
    out[OUT_RESULT] = output.relinquish() ;
  }
}
