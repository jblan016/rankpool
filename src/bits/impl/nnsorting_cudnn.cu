// @file nnsorting_blas.cu
// @brief Sorting block CuDNN-based implementation.
// @author Andrea Vedaldi

/*
 Copyright (C) 2015-16 Andrea Vedaldi.
 All rights reserved.

 This file is part of the VLFeat library and is made available under
 the terms of the BSD license (see the COPYING file).
 */

#if !defined(ENABLE_GPU) | !defined(ENABLE_CUDNN)
#error "nnsorting_cudnn.hpp cannot be compiled without GPU and CUDNN support."
#endif

#include "nnsorting_cudnn.hpp"
#include "cudnnhelper.hpp"
#include "../datacu.hpp"
#include <assert.h>

using namespace vl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = context.setError(context.getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
goto done ; \
} }

/* ---------------------------------------------------------------- */
/*                                         nnsorting_cudnn::forward */
/* ---------------------------------------------------------------- */


namespace vl { namespace impl {


  template<vl::DataType dataType>
  vl::ErrorCode
  nnsorting_cudnn<dataType>::forward(Context& context,
                                     Tensor output,
                                     Tensor data,
                                     SortingMethod method,
                                     int sortHeight, int sortWidth,
                                     int strideY, int strideX,
                                     int padTop, int padBottom,
                                     int padLeft, int padRight)
  {
    assert(output) ;
    assert(data) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, dataDesc ;
    cudnnSortingDescriptor_t sortingDesc ;
    bool outputDescInitialized = false ;
    bool dataDescInitialized = false ;
    bool sortingDescInitialized = false ;

    if (padLeft != padRight) return vl::VLE_Unsupported ;
    if (padTop != padBottom) return vl::VLE_Unsupported ;

    if (method == vlSortingAverage && (padLeft > 0 | padRight > 0)) {
      /* This seems like a bug in CUDNN? */
      return vl::VLE_Unsupported ;
    }

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::id ;
    vl::DataType dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     output.getSize(), // sizes
                                     output.getDepth(),
                                     output.getWidth(),
                                     output.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     data.getSize(),
                                     data.getDepth(),
                                     data.getWidth(),
                                     data.getHeight())) ;

    CHECK(cudnnCreateSortingDescriptor(&sortingDesc)) ;
    sortingDescInitialized = true ;
    CHECK(cudnnSetSorting2dDescriptor(sortingDesc,
                                      (method == vl::vlSortingAverage) ? CUDNN_SORTING_AVERAGE_COUNT_INCLUDE_PADDING : CUDNN_SORTING_MAX,
                                      IF_CUDNN_GE5(HIPDNN_NOT_PROPAGATE_NAN COMMA)
                                      sortWidth, sortHeight,
                                      padLeft, padTop,
                                      strideX, strideY)) ;

    // Perform convolution for each filter group
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      CHECK(cudnnSortingForward(handle,
                                sortingDesc,
                                &alpha,
                                dataDesc, data.getMemory(),
                                &beta,
                                outputDesc, output.getMemory())) ;
    }

    /* cleanup */
  done:
    if (sortingDescInitialized) { cudnnDestroySortingDescriptor(sortingDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return context.passError(error, "nnsorting_cudnn::forward") ;
  }

  /* ---------------------------------------------------------------- */
  /*                                        nnsorting_cudnn::backward */
  /* ---------------------------------------------------------------- */

  template<vl::DataType dataType>
  vl::ErrorCode
  nnsorting_cudnn<dataType>::backward(Context& context,
                                      Tensor derData,
                                      Tensor data,
                                      Tensor output,
                                      Tensor derOutput,
                                      vl::SortingMethod method,
                                      int sortHeight, int sortWidth,
                                      int strideY, int strideX,
                                      int padTop, int padBottom,
                                      int padLeft, int padRight)
  {
    assert(derData) ;
    assert(data) ;
    assert(output) ;
    assert(derOutput) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, dataDesc ;
    cudnnSortingDescriptor_t sortingDesc ;
    bool outputDescInitialized = false ;
    bool dataDescInitialized = false ;
    bool sortingDescInitialized = false ;

    if (padLeft != padRight) return vl::VLE_Unsupported ;
    if (padTop != padBottom) return vl::VLE_Unsupported ;

    if (method == vlSortingAverage && (padLeft > 0 | padRight > 0)) {
      /* This seems like a bug in CuDNN? */
      return vl::VLE_Unsupported ;
    }

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::id ;
    vl::DataType dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     output.getSize(), // sizes
                                     output.getDepth(),
                                     output.getWidth(),
                                     output.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     data.getSize(),
                                     data.getDepth(),
                                     data.getWidth(),
                                     data.getHeight())) ;

    CHECK(cudnnCreateSortingDescriptor(&sortingDesc)) ;
    sortingDescInitialized = true ;
    CHECK(cudnnSetSorting2dDescriptor(sortingDesc,
                                      (method == vl::vlSortingAverage) ? CUDNN_SORTING_AVERAGE_COUNT_INCLUDE_PADDING : CUDNN_SORTING_MAX,
                                      IF_CUDNN_GE5(HIPDNN_NOT_PROPAGATE_NAN COMMA)
                                      sortWidth, sortHeight,
                                      padLeft, padTop,
                                      strideX, strideY)) ;

    // Perform convolution for each filter group
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      CHECK(cudnnSortingBackward(handle,
                                 sortingDesc,
                                 &alpha,
                                 outputDesc, (type const*)output.getMemory(),
                                 outputDesc, (type const*)derOutput.getMemory(),
                                 dataDesc, (type const*)data.getMemory(),
                                 &beta,
                                 dataDesc, (type*)derData.getMemory())) ;
    }

    /* cleanup */
  done:
    if (sortingDescInitialized) { cudnnDestroySortingDescriptor(sortingDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return context.passError(error, __func__) ;
  }
  
} }

// Instantiations
template struct vl::impl::nnsorting_cudnn<vl::VLDT_Float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::nnsorting_cudnn<vl::VLDT_Double> ;
#endif



